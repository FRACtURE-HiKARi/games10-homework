#include "hip/hip_runtime.h"
#include "Scene.hpp"
#include "CudaRender.h"
#include "CudaHelper.hpp"
#include <thread>
#include <chrono>

#define MAX_DEPTH 64
#define MAX_TRIANGLES 1024
#define P_RR 0.8
#define COMP(X) X.x, X.y, X.z
__device__ float spp;
__device__ float scale_d;
__device__ float imageAspectRatio_d;
__device__ int width_d;
__device__ int height_d;
__device__ Vector3f* eye_pos_d;
Vector3f* framebuffer;
Triangle_d* triangles;
Triangle* triangles_h;
__device__ int num_triangles = 0;
int num_pixels;

inline void printLastErr()
{
    hipError_t launchErr = hipGetLastError();
    if (launchErr != hipSuccess) {
        printf("Last Error: %s\n", hipGetErrorString(launchErr));
    }
    else {
        printf("Success. \n");
    }
}

void init_objects(const Scene& scene)
{
    triangles_h = (Triangle*)malloc(sizeof(Triangle) * MAX_TRIANGLES);
    int i = 0;
    for (auto o: scene.objects)
    {
        for (auto t: ((MeshTriangle*)o)->triangles)
        {
            if (i < MAX_TRIANGLES)
            {
                triangles_h[i++] = t;
            }
        }
    }
    hipMalloc(&triangles, i * sizeof(Triangle_d));
    hipMemcpyToSymbol(HIP_SYMBOL(num_triangles), &i, sizeof(int));
    printf("Num Triangles: %d\n", i);
    for (int j = 0; j < i; j++)
    {
        triangleToDevice(&triangles[j], triangles_h[j]);
    }
}

void init_memory(const Scene& scene, int spp)
{
    float scale = tan(deg2rad(scene.fov * 0.5));
    float imageAspectRatio = scene.width / (float)scene.height;
    hipMemcpyToSymbol(HIP_SYMBOL(scale_d), &scale, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(imageAspectRatio_d), &imageAspectRatio, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(width_d), &scene.width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(height_d), &scene.height, sizeof(int));
    
    hipMalloc(&framebuffer, num_pixels * sizeof(Vector3f));
    hipMemset(framebuffer, 0, num_pixels * sizeof(Vector3f));
    init_objects(scene);
}

void clean_up()
{
    free(triangles_h);
    hipFree(framebuffer);
    hipFree(triangles);
    Vector3f* tmp;
    hipMemcpyFromSymbol(&tmp, HIP_SYMBOL(eye_pos_d), sizeof(Vector3f*));
    hipFree(tmp);
}

__device__ Intersection_d getClosest(const Ray& ray, Triangle_d* ts, int depth)
{
    Intersection_d result;
    for (int i = 0; i < num_triangles; i++)
    {
        auto t = ts[i];
        Intersection_d inter = t.getIntersection(ray);
        if (inter.happened && inter.distance < result.distance)
        {
            result = inter;
        }
    }
    return result;
}

__device__ Intersection_d getClosest(const Ray &ray, Triangle_d* ts)
{
    return getClosest(ray, ts, 0);
}

// Implementation of PathTracing in CUDA
__device__ Vector3f trace(const Ray& ray, Triangle_d* ts, hiprandState* state)
{
    int depth = 0;
    Vector3f stack_dir[MAX_DEPTH];
    Vector3f stack_multiplier[MAX_DEPTH];

    Ray currentRay = ray;
    while (depth < MAX_DEPTH)
    {
        stack_dir[depth] = Vector3f(0);
        stack_multiplier[depth] = Vector3f(0);
        Intersection_d inter = getClosest(currentRay, ts);
        if (!inter.happened){
            // return Vector3f(0);
            break;
        }
        if (inter.m->hasEmission()){
            stack_dir[depth] = inter.m->getEmission();
            break;
        }
        Vector3f p = inter.coords;
        Material* m = inter.m;
        float pdf_light;
        Intersection inter_light;
        sampleLight(ts, num_triangles, inter_light, pdf_light, state);
        // test if blocked
        Vector3f x = inter_light.coords;
        Vector3f ws = (x - p).normalized();
        Vector3f wo = currentRay.direction;
        Vector3f N = inter.normal;
        Ray dir_ray(p, ws);
        Intersection_d block_test = getClosest(dir_ray, ts);
        if (block_test.distance - (x - p).norm() > -0.005)
        {
            Vector3f NN = inter_light.normal;
            stack_dir[depth] = inter_light.emit * m->eval(wo, ws, N) * dotProduct(ws, N) * dotProduct(-ws, NN) / dotProduct(x-p, x-p) / pdf_light;
        }
        if (hiprand_uniform(state) > P_RR){
            break;
        }
        
        //Vector3f wi = m->sample(wo, N);
        Vector3f wi = sampleMaterial(m, -wo, N, state);
        Ray indir_ray(p, wi);
        Intersection_d nonemit_inter = getClosest(indir_ray, ts);
        if (nonemit_inter.happened && !nonemit_inter.m->hasEmission())
        {
            //L_indir = trace(indir_ray, ts, depth + 1, state) * m->eval(wo, wi, N) * dotProduct(wi, N) / m->pdf(wo, wi, N) / P_RR;
            stack_multiplier[depth] = m->eval(wo, wi, N) * dotProduct(wi, N) / m->pdf(wo, wi, N) / P_RR;
            currentRay = indir_ray;
            depth += 1;
            continue;
        }
    }
    // accumulate all lights
    Vector3f result = stack_dir[depth--];
    while (depth >= 0)
    {
        result = result * stack_multiplier[depth] + stack_dir[depth];
        depth -= 1;
    }
    return result;
}

// kernel thread
__global__
void CUDA_PT(Vector3f* fb, Triangle_d* ts, int spp, hiprandState* states)
{
    if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 & blockIdx.y == 0)
    {
        eye_pos_d = new Vector3f(278.0f, 273.0f, -800.0f);
    }
    int xi = threadIdx.x + blockIdx.x * blockDim.x;
    int yi = threadIdx.y + blockIdx.y * blockDim.y;
    int tid = xi + width_d * yi;
    float x = (2 * (xi + 0.5) / (float)width_d - 1) * imageAspectRatio_d * scale_d;
    float y = (1 - 2 * (yi + 0.5) / (float)height_d) * scale_d;
    hiprand_init(1234, tid, 0, &states[tid]);
    Vector3f result;
    Vector3f dir = normalize(Vector3f(-x, y, 1));
    Ray ray(*eye_pos_d, dir);
    for (int i = 0; i < spp; i++)
    {
        result += trace(ray, ts, &states[tid]) / (float)spp;
    }
    fb[tid] = result;
}

void gpuWaitThread() {hipDeviceSynchronize();}

#define BLOCK_DIM 32
void cudaRender(Vector3f* fb_h, const Scene& scene, int spp)
{
    dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
    dim3 gridDim(
        (scene.width + BLOCK_DIM - 1) / BLOCK_DIM,
        (scene.height + BLOCK_DIM - 1) / BLOCK_DIM
    );
    num_pixels = scene.width * scene.height;
    printf("Init memory...\n");
    init_memory(scene, spp);
    printLastErr();

    hiprandState* states_d;
    hipMalloc(&states_d, num_pixels * sizeof(hiprandState));

    printf("Starting Kernel\n");
    CUDA_PT<<<gridDim, blockDim>>>(framebuffer, triangles, spp, states_d);
    printLastErr();
    printf("Waiting for sync\n");
    hipDeviceSynchronize();
    printLastErr();
    hipMemcpy(fb_h, framebuffer, sizeof(Vector3f) * num_pixels, hipMemcpyDeviceToHost);
}